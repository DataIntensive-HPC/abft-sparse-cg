#include "hip/hip_runtime.h"
#include "HIPContext.h"
#include "CUDAecc.h"

#include <cstdio>
#include <cstdlib>
#include <cstring>

__global__ void inject_bitflip_val(
  const uint32_t bit, //vector size
  const uint32_t index,
  double * values)
{
  printf("*** flipping bit %u of value at index %u ***\n", bit, index);
	*((uint64_t*)values+index) ^= 0x1 << (bit % 32);
}

__global__ void inject_bitflip_col(
  const uint32_t bit, //vector size
  const uint32_t index,
  uint32_t * values)
{
  printf("*** flipping bit %u of column at index %u ***\n", bit, index);
  values[index] ^= 0x1 << (bit % 32);
}

template <uint32_t blockSize, uint32_t items_per_work_item, uint32_t items_per_work_group>
__global__ void dot_product_kernel(
  const unsigned int N,
  const double * __restrict__ a,
  const double * __restrict__ b,
  double * __restrict__ result)
{
  __shared__ double partial_result[blockSize];

  const uint32_t local_id = threadIdx.x;
  const uint32_t group_size = blockDim.x;
  const uint32_t group_id = blockIdx.x;

  double ret = 0.0;
  double tmp;
  uint32_t offset = group_id * items_per_work_group + local_id;
  for (uint32_t i = 0; i < items_per_work_item; i++, offset += group_size)
  {
    uint8_t in_range = offset < N;
    uint32_t local_offset = in_range ? offset : 0;
    tmp = a[local_offset] * b[local_offset];
    ret += in_range ? tmp : 0.0;
  }

  partial_result[local_id] = ret;

  //do a reduction
  for(uint32_t step = blockSize >> 1; step > 0; step>>=1)
  {
		__syncthreads();
    if(local_id < step)
    {
      partial_result[local_id] += partial_result[local_id + step];
    }
  }

	__syncthreads();
  if(local_id == 0)
  {
  	result[group_id] = partial_result[0];
  }
}

template <uint32_t items_per_work_item, uint32_t items_per_work_group>
__global__ void calc_p_kernel(
  const uint32_t N, //vector size
  const double beta,
  const double * __restrict__ r,
  double * __restrict__ p)
{
  const uint32_t local_id = threadIdx.x;
  const uint32_t group_size = blockDim.x;
  const uint32_t group_id = blockIdx.x;

  uint32_t offset = group_id * items_per_work_group + local_id;
  for (uint32_t i = 0; i < items_per_work_item && offset < N; i++, offset += group_size)
  {
    p[offset] = fma(beta, p[offset], r[offset]);
  }

}

template <uint32_t blockSize, uint32_t items_per_work_item, uint32_t items_per_work_group>
__global__ void calc_xr_kernel(
  const uint32_t N, //vector size
  const double alpha,
  const double * __restrict__ p,
  const double * __restrict__ w,
  double * __restrict__ x,
  double * __restrict__ r,
  double * __restrict__ result)
{
  __shared__ double partial_result[blockSize];

  const uint32_t local_id = threadIdx.x;
  const uint32_t group_size = blockDim.x;
  const uint32_t group_id = blockIdx.x;

  double ret = 0.0;

  const uint32_t offset = group_id * items_per_work_group + local_id;

  uint32_t j = offset;
  for (uint32_t i = 0; i < items_per_work_item && j < N; i++, j += group_size)
  {
    x[j] = fma(alpha, p[j], x[j]);
  }

  j = offset;
  for (uint32_t i = 0; i < items_per_work_item && j < N; i++, j += group_size)
  {
    r[j] = fma(-alpha, w[j], r[j]);
    ret = fma(r[j], r[j], ret);
  }
  partial_result[local_id] = ret;
  //do a reduction
  for(uint32_t step = group_size >> 1; step > 1; step>>=1)
  {
    __syncthreads();
    if(local_id < step)
    {
      partial_result[local_id] += partial_result[local_id + step];
    }
  }
  //store result in a global array
  __syncthreads();
  if(local_id == 0)
  {
    result[group_id] = partial_result[0] + partial_result[1];
  }
}


//CSR_SCALAR TECHNIQUE
template <FT_Type ftType>
__global__ void spmv_scalar_kernel(
  const uint32_t N, //vector size
  const uint32_t * __restrict__ mat_rows,
  uint32_t * __restrict__ mat_cols,
  double * __restrict__ mat_values,
  const double * __restrict__ vec,
  double * __restrict__ result,
  const uint32_t nnz)
{
  const uint32_t global_id = threadIdx.x + blockDim.x * blockIdx.x;
  if(global_id < N)
  {
    uint32_t start = mat_rows[global_id];
    uint32_t end   = mat_rows[global_id+1];

	if(ftType == CONSTRAINTS)
	{
    if(end > nnz)
    {
      printf("row size constraint violated for row %u\n", global_id);
      __threadfence();
  		asm("trap;");
    }
    else if(end < start)
    {
      printf("row order constraint violated for row %u\n", global_id);
      __threadfence();
  		asm("trap;");
    }
	}

    // initialize local sum
    double tmp = 0;
    csr_element element;
    uint32_t syndrome;
    // accumulate local sums
    for(uint32_t i = start; i < end; i++)
    {
      uint32_t col = mat_cols[i];
      switch(ftType)
      {
      	case CONSTRAINTS:
		      if(col >= N)
		      {
	          printf("column size constraint violated at index %u\n", i);
			      __threadfence();
			  		asm("trap;");
		      }
		      else if(i < end-1 && mat_cols[i+1] <= col)
		      {
	          printf("column order constraint violated at index %u\n", i);
			      __threadfence();
			  		asm("trap;");
		      }
      	break; //CONSTRAINTS
				case SED:
		      element.value  = mat_values[i];
		      element.column = col;
		      // Check overall parity bit
		      if(cu_ecc_compute_overall_parity(element))
		      {
	          printf("[ECC] error detected at index %u\n", i);
			      __threadfence();
			  		asm("trap;");
		      }
		      // Mask out ECC from high order column bits
		      element.column &= 0x00FFFFFF;
		      col = element.column;
		    break; //SED
				case SEC7:
		      element.value  = mat_values[i];
		      element.column = col;
		      // Check ECC
		      uint32_t syndrome = cu_ecc_compute_col8(element);
		      if(syndrome)
		      {
		        // Unflip bit
		        uint32_t bit = cu_ecc_get_flipped_bit_col8(syndrome);
		        ((uint*)(&element))[bit/32] ^= 0x1 << (bit % 32);
		        mat_cols[i] = element.column;
		        mat_values[i] = element.value;
		        printf("[ECC] corrected bit %u at index %u\n", bit, i);
		      }

		      // Mask out ECC from high order column bits
		      element.column &= 0x00FFFFFF;
		      col = element.column;
				break; //SEC7
				case SEC8:
		      element.value  = mat_values[i];
		      element.column = col;
		      // Check overall parity bit
		      if(cu_ecc_compute_overall_parity(element))
		      {
		        // Compute error syndrome from hamming bits
		        syndrome = cu_ecc_compute_col8(element);
		        if(syndrome)
		        {
		          // Unflip bit
		          uint32_t bit = cu_ecc_get_flipped_bit_col8(syndrome);
		          ((uint*)(&element))[bit/32] ^= 0x1 << (bit % 32);
		          printf("[ECC] corrected bit %u at index %u\n", bit, i);
		        }
		        else
		        {
		          // Correct overall parity bit
		          element.column ^= 0x1 << 24;
		          printf("[ECC] corrected overall parity bit at index %u\n", i);
		        }

		        mat_cols[i] = element.column;
		        mat_values[i] = element.value;
		      }
		      // Mask out ECC from high order column bits
		      element.column &= 0x00FFFFFF;
		      col = element.column;
				break; //SEC8
				case SECDED:
		      element.value  = mat_values[i];
		      element.column = col;
		      // Check parity bits
		      uint32_t overall_parity = cu_ecc_compute_overall_parity(element);
		      syndrome = cu_ecc_compute_col8(element);
		      if(overall_parity)
		      {
		        if(syndrome)
		        {
		          // Unflip bit
		          uint32_t bit = cu_ecc_get_flipped_bit_col8(syndrome);
		          ((uint*)(&element))[bit/32] ^= 0x1 << (bit % 32);
		          printf("[ECC] corrected bit %u at index %d\n", bit, i);
		        }
		        else
		        {
		          // Correct overall parity bit
		          element.column ^= 0x1 << 24;
		          printf("[ECC] corrected overall parity bit at index %d\n", i);
		        }

		        mat_cols[i] = element.column;
		        mat_values[i] = element.value;
		      }
		      else
		      {
		        if(syndrome)
		        {
		          // Overall parity fine but error in syndrom
		          // Must be double-bit error - cannot correct this
            	printf("[ECC] double-bit error detected\n");
				      __threadfence();
				  		asm("trap;");
		        }
		      }
		      // Mask out ECC from high order column bits
		      element.column &= 0x00FFFFFF;
		      col = element.column;
				break;
			}
      tmp = fma(mat_values[i], vec[col], tmp);
    }
    result[global_id] = tmp;
  }
}

void HIPContext::generate_ecc_bits(csr_element& element)
{
}

HIPContext::HIPContext(FT_Type type)
{
  ftType = type;

  k_inject_bitflip_val = new cuda_kernel;
  k_inject_bitflip_val->first_run = 1;

  k_inject_bitflip_col = new cuda_kernel;
  k_inject_bitflip_col->first_run = 1;

  k_dot_product = new cuda_kernel;
  k_dot_product->first_run = 1;

  k_calc_xr = new cuda_kernel;
  k_calc_xr->first_run = 1;

  k_calc_p = new cuda_kernel;
  k_calc_p->first_run = 1;

  k_spmv = new cuda_kernel;
  k_spmv->first_run = 1;
}

HIPContext::~HIPContext()
{
  delete[] h_dot_product_partial;
  delete[] h_calc_xr_partial;

  cudaCheck(hipFree(d_dot_product_partial));
  cudaCheck(hipFree(d_calc_xr_partial));
}

cg_matrix* HIPContext::create_matrix(const uint32_t *columns,
                                     const uint32_t *rows,
                                     const double *values,
                                     int N, int nnz)
{
  cg_matrix* M = new cg_matrix;
  M->N      = N;
  M->nnz    = nnz;
  //allocate buffers on the device
  cudaCheck(hipMalloc((void**)&M->cols, sizeof(uint32_t) * nnz));
  cudaCheck(hipMalloc((void**)&M->rows, sizeof(uint32_t) * (N+1)));
  cudaCheck(hipMalloc((void**)&M->values, sizeof(double) * nnz));

  //allocate temp memory which is then copied to the device
  uint32_t *h_cols   = new uint32_t[nnz];
  uint32_t *h_rows   = new uint32_t[N+1];
  double   *h_values = new double[nnz];

  uint32_t next_row = 0;
  for (int i = 0; i < nnz; i++)
  {
    csr_element element;
    element.column = columns[i];
    element.value  = values[i];

    generate_ecc_bits(element);

    h_cols[i]   = element.column;
    h_values[i] = element.value;

    while (next_row <= rows[i])
    {
      h_rows[next_row++] = i;
    }
  }
  h_rows[N] = nnz;
  cudaCheck(hipMemcpy(M->cols, h_cols, sizeof(uint32_t) * nnz, hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(M->rows, h_rows, sizeof(uint32_t) * (N+1), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(M->values, h_values, sizeof(double) * nnz, hipMemcpyHostToDevice));

  //clean up temp buffers
  delete[] h_cols;
  delete[] h_rows;
  delete[] h_values;

  return M;
}

void HIPContext::destroy_matrix(cg_matrix *mat)
{
  cudaCheck(hipFree(mat->cols));
  cudaCheck(hipFree(mat->rows));
  cudaCheck(hipFree(mat->values));
  delete mat;
}

cg_vector* HIPContext::create_vector(int N)
{
  cg_vector *result = new cg_vector;
  result->N = N;
  cudaCheck(hipMalloc((void**)&result->data, sizeof(double) * N));
  return result;
}

void HIPContext::destroy_vector(cg_vector *vec)
{
  cudaCheck(hipFree(vec->data));
  delete vec;
}

double* HIPContext::map_vector(cg_vector *v)
{
  double* h = new double[v->N];
  cudaCheck(hipMemcpy(h, v->data, sizeof(double) * v->N, hipMemcpyDeviceToHost));
  return h;
}

void HIPContext::unmap_vector(cg_vector *v, double *h)
{
  cudaCheck(hipMemcpy(v->data, h, sizeof(double) * v->N, hipMemcpyHostToDevice));
  delete[] h;
}

void HIPContext::copy_vector(cg_vector *dst, const cg_vector *src)
{
  cudaCheck(hipMemcpy(dst->data, src->data, sizeof(double) * dst->N, hipMemcpyDeviceToDevice));
}

double HIPContext::dot(const cg_vector *a, const cg_vector *b)
{
  if(k_dot_product->first_run){
    HIPContext::setup_cuda_kernel(k_dot_product, DOT_PRODUCT_KERNEL_ITEMS_PER_WORK_ITEM, DOT_PRODUCT_KERNEL_WG, a->N);
#if VECTOR_SUM_METHOD_USE == VECTOR_SUM_NO_PINNED
    cudaCheck(hipMalloc((void**)&d_dot_product_partial, sizeof(double) * k_dot_product->ngroups));
    h_dot_product_partial = new double[k_dot_product->ngroups];
#elif VECTOR_SUM_METHOD_USE == VECTOR_SUM_PINNED
    d_dot_product_partial = clCreateBuffer(ocl_context, CL_MEM_ALLOC_HOST_PTR, sizeof(double) * k_dot_product->ngroups, NULL, &err);
#endif
  }

  dot_product_kernel<DOT_PRODUCT_KERNEL_WG,
  									 DOT_PRODUCT_KERNEL_ITEMS_PER_WORK_ITEM,
  									 DOT_PRODUCT_KERNEL_WG*DOT_PRODUCT_KERNEL_ITEMS_PER_WORK_ITEM>
	  								<<<k_dot_product->ngroups,DOT_PRODUCT_KERNEL_WG>>>
	  								(a->N, a->data, b->data, d_dot_product_partial);

  return HIPContext::sum_vector(d_dot_product_partial, h_dot_product_partial, k_dot_product->ngroups);
}

double HIPContext::calc_xr(cg_vector *x, cg_vector *r,
                           const cg_vector *p, const cg_vector *w,
                           double alpha)
{
  if(k_calc_xr->first_run){
    HIPContext::setup_cuda_kernel(k_calc_xr, CALC_XR_KERNEL_ITEMS_PER_WORK_ITEM, CALC_XR_KERNEL_WG, x->N);
#if VECTOR_SUM_METHOD_USE == VECTOR_SUM_NO_PINNED
    cudaCheck(hipMalloc((void**)&d_calc_xr_partial, sizeof(double) * k_calc_xr->ngroups));
    h_calc_xr_partial = new double[k_calc_xr->ngroups];
#elif VECTOR_SUM_METHOD_USE == VECTOR_SUM_PINNED
    d_calc_xr_partial = clCreateBuffer(ocl_context, CL_MEM_ALLOC_HOST_PTR, sizeof(double) * k_calc_xr->ngroups, NULL, &err);
#endif
  }

	calc_xr_kernel<CALC_XR_KERNEL_WG,
								 CALC_XR_KERNEL_ITEMS_PER_WORK_ITEM,
								 CALC_XR_KERNEL_WG*CALC_XR_KERNEL_ITEMS_PER_WORK_ITEM>
								<<<k_calc_xr->ngroups,CALC_XR_KERNEL_WG>>>
							  (x->N, alpha, p->data, w->data, x->data, r->data, d_calc_xr_partial);
  return HIPContext::sum_vector(d_calc_xr_partial, h_calc_xr_partial, k_calc_xr->ngroups);
}

void HIPContext::calc_p(cg_vector *p, const cg_vector *r, double beta)
{
  if(k_calc_p->first_run){
    HIPContext::setup_cuda_kernel(k_calc_p, CALC_P_KERNEL_ITEMS_PER_WORK_ITEM, CALC_P_KERNEL_WG, p->N);
  }

	calc_p_kernel<CALC_P_KERNEL_ITEMS_PER_WORK_ITEM,
	              CALC_P_KERNEL_ITEMS_PER_WORK_ITEM*CALC_P_KERNEL_WG>
	             <<<k_calc_p->ngroups,CALC_P_KERNEL_WG>>>
	             (p->N, beta, r->data, p->data);
}

void HIPContext::spmv(const cg_matrix *mat, const cg_vector *vec,
                      cg_vector *result)
{

  if(k_spmv->first_run){
    size_t total_work = mat->N;
#if SPMV_METHOD == SPMV_VECTOR

    const int nnz_per_row = mat->nnz / mat->N;

    if (nnz_per_row <=  2)
    {
      _SPMV_THREADS_PER_VECTOR = 2;
    }
    else if (nnz_per_row <=  4)
    {
      _SPMV_THREADS_PER_VECTOR = 4;
    }
    else if (nnz_per_row <=  8)
    {
      _SPMV_THREADS_PER_VECTOR = 8;
    }
    else if (nnz_per_row <= 16)
    {
      _SPMV_THREADS_PER_VECTOR = 16;
    }
    else if (nnz_per_row <= 32)
    {
      _SPMV_THREADS_PER_VECTOR = 32;
    }
    else
    {
      _SPMV_THREADS_PER_VECTOR = 64;
    }

    _SPMV_VECTORS_PER_BLOCK  = SPMV_KERNEL_WG / _SPMV_THREADS_PER_VECTOR;
    total_work = mat->nnz;
#endif
    HIPContext::setup_cuda_kernel(k_spmv, SPMV_KERNEL_ITEMS_PER_WORK_ITEM, SPMV_KERNEL_WG, total_work);
  }
#if SPMV_METHOD == SPMV_SCALAR
  switch(ftType)
  {
  	case NONE:
  		spmv_scalar_kernel<NONE>
	                  		<<<k_spmv->ngroups,SPMV_KERNEL_WG>>>
	                  		(mat->N, mat->rows, mat->cols, mat->values, vec->data, result->data, mat->nnz);
		break;
  	case CONSTRAINTS:
  		spmv_scalar_kernel<CONSTRAINTS>
	                  		<<<k_spmv->ngroups,SPMV_KERNEL_WG>>>
	                  		(mat->N, mat->rows, mat->cols, mat->values, vec->data, result->data, mat->nnz);
		break;
  	case SED:
  		spmv_scalar_kernel<SED>
	                  		<<<k_spmv->ngroups,SPMV_KERNEL_WG>>>
	                  		(mat->N, mat->rows, mat->cols, mat->values, vec->data, result->data, mat->nnz);
		break;
  	case SEC7:
  		spmv_scalar_kernel<SEC7>
	                  		<<<k_spmv->ngroups,SPMV_KERNEL_WG>>>
	                  		(mat->N, mat->rows, mat->cols, mat->values, vec->data, result->data, mat->nnz);
		break;
  	case SEC8:
  		spmv_scalar_kernel<SEC8>
	                  		<<<k_spmv->ngroups,SPMV_KERNEL_WG>>>
	                  		(mat->N, mat->rows, mat->cols, mat->values, vec->data, result->data, mat->nnz);
		break;
  	case SECDED:
  		spmv_scalar_kernel<SECDED>
	                  		<<<k_spmv->ngroups,SPMV_KERNEL_WG>>>
	                  		(mat->N, mat->rows, mat->cols, mat->values, vec->data, result->data, mat->nnz);
		break;
	}
#elif SPMV_METHOD == SPMV_VECTOR

#endif
}

double HIPContext::sum_vector(double * d_buffer, double * h_buffer, const uint32_t N)
{
  //sum the vector in the kernel
	double result = 0;
#if VECTOR_SUM_METHOD_USE == VECTOR_SUM_NO_PINNED
	cudaCheck(hipMemcpy(h_buffer, d_buffer, sizeof(double) * N, hipMemcpyDeviceToHost));
#elif VECTOR_SUM_METHOD_USE == VECTOR_SUM_PINNED
  h_buffer = (double *) clEnqueueMapBuffer(ocl_queue, d_buffer, CL_TRUE, CL_MAP_READ, 0, sizeof(cl_double) * N, 0, NULL, NULL, &err);
  if (CL_SUCCESS != err) DIE("OpenCL error %d whilst mapping pinned memory", err);
#endif

  for(uint32_t i = 0; i < N; i++){
    result += h_buffer[i];
  }

#if VECTOR_SUM_METHOD_USE == VECTOR_SUM_PINNED
  err = clEnqueueUnmapMemObject(ocl_queue, d_buffer, h_buffer, 0, NULL, NULL);
  if (CL_SUCCESS != err) DIE("OpenCL error %d whilst unmapping pinnned memory", err);
#endif
  return result;

}

void HIPContext::setup_cuda_kernel(cuda_kernel* kernel, const size_t items_per_work_item, const size_t group_size, const size_t total_work)
{
  kernel->group_size = group_size;
  kernel->items_per_work_item = items_per_work_item;
  kernel->ngroups = ceil((float)total_work / (float)(group_size * items_per_work_item));
  kernel->global_size = group_size * kernel->ngroups;
  kernel->first_run = 0;
}

void HIPContext::inject_bitflip(cg_matrix *mat, BitFlipKind kind, int num_flips)
{
  uint32_t index = rand() % mat->nnz;

  uint32_t start = 0;
  uint32_t end   = 96;

  if (kind == VALUE)
    end = 64;
  else if (kind == INDEX)
    start = 64;

  for (int i = 0; i < num_flips; i++)
  {
    uint32_t bit = (rand() % (end-start)) + start;
    if (bit < 64)
    {
			inject_bitflip_val<<<1,1>>>(bit, index, mat->values);
    }
    else
    {
      bit = bit - 64;
			inject_bitflip_col<<<1,1>>>(bit, index, mat->cols);
    }
  }
}

void CUDAContext_SED::generate_ecc_bits(csr_element& element)
{
  element.column |= ecc_compute_overall_parity(element) << 31;
}

void CUDAContext_SEC7::generate_ecc_bits(csr_element& element)
{
  element.column |= ecc_compute_col8(element);
}

void CUDAContext_SEC8::generate_ecc_bits(csr_element& element)
{
  element.column |= ecc_compute_col8(element);
  element.column |= ecc_compute_overall_parity(element) << 24;
}

void CUDAContext_SECDED::generate_ecc_bits(csr_element& element)
{
  element.column |= ecc_compute_col8(element);
  element.column |= ecc_compute_overall_parity(element) << 24;
}

namespace
{
  static CGContext::Register<HIPContext> A("cuda", "none");
  static CGContext::Register<CUDAContext_Constraints> B("cuda", "constraints");
  static CGContext::Register<CUDAContext_SED> C("cuda", "sed");
  static CGContext::Register<CUDAContext_SEC7> D("cuda", "sec7");
  static CGContext::Register<CUDAContext_SEC8> E("cuda", "sec8");
  static CGContext::Register<CUDAContext_SECDED> F("cuda", "secded");
}